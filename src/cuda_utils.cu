#include "cuda_utils.h"

void *cuda_alloc(size_t size)
{
    void *ptr;
    CHECK_CUDA(hipMalloc(&ptr, size));
    return ptr;
}

void cuda_free(void *ptr)
{
    CHECK_CUDA(hipFree(ptr));
}

void copy_to_device(void *dst, const void *src, size_t size)
{
    CHECK_CUDA(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void copy_to_host(void *dst, const void *src, size_t size)
{
    CHECK_CUDA(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}